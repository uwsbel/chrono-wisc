#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for box geometries
//
// =============================================================================

#ifdef _WIN32
#ifndef NOMINMAX
#define NOMINMAX
#endif
#endif

#include "chrono_sensor/optix/shaders/device_utils.h"

static __device__ float3 box_normal(float t, float3 t0, float3 t1) {
    float3 normal_pos = make_float3(t == t0.x ? 1 : 0, t == t0.y ? 1 : 0, t == t0.z ? 1 : 0);
    float3 normal_neg = make_float3(t == t1.x ? 1 : 0, t == t1.y ? 1 : 0, t == t1.z ? 1 : 0);
    return normal_neg - normal_pos;
}

extern "C" __global__ void __intersection__box_intersect() {
    const float3 ray_orig = optixGetObjectRayOrigin();
    const float3 ray_dir = optixGetObjectRayDirection();
    const float ray_tmin = optixGetRayTmin();
    const float ray_tmax = optixGetRayTmax();

  /*  RayType raytype = (RayType)optixGetPayload_2();
    if (raytype == RayType::LASER_SAMPLE_RAY_TYPE) {
        printf("ray_orig: (%f, %f, %f), ray_dir: (%f, %f, %f), ray_tmin: %f, ray_tmax: %f\n", ray_orig.x, ray_orig.y,
               ray_orig.z, ray_dir.x, ray_dir.y, ray_dir.z, ray_tmin, ray_tmax);
   } */ 


    // calculate potential intersections with the box
    float3 t0 = (make_float3(-.5f) - ray_orig) / ray_dir;
    float3 t1 = (make_float3(.5f) - ray_orig) / ray_dir;
    float3 near = fminf(t0, t1);
    float3 far = fmaxf(t0, t1);
    // dist_near and dist_far are the distances to the potential intsection points
    float dist_near = fmaxf(near);
    float dist_far = fminf(far);

    // check if near is less than far
    if (dist_near <= dist_far) {
        float3 p = make_float3(0);

        if (dist_near > ray_tmin && dist_near < ray_tmax) {
            float3 shading_normal = box_normal(dist_near, t0, t1);
            float2 texcoord;
            float3 tangent_vector;
            p = ray_orig + dist_near * ray_dir;

            if (abs(shading_normal.x) > 0.5) {
                texcoord = make_float2((p.y + 0.5), (p.z + 0.5) * shading_normal.x);
                tangent_vector = make_float3(0, 1, 0);
            } else if (abs(shading_normal.y) > 0.5) {
                texcoord = make_float2((p.x + 0.5), -(p.z + 0.5) * shading_normal.y);
                tangent_vector = make_float3(1, 0, 0);
            } else {
                texcoord = make_float2((p.x + 0.5), (p.y + 0.5) * shading_normal.z);
                tangent_vector = make_float3(1, 0, 0);
            }

            optixReportIntersection(
                dist_near, 0, reinterpret_cast<unsigned int&>(shading_normal.x),
                reinterpret_cast<unsigned int&>(shading_normal.y), reinterpret_cast<unsigned int&>(shading_normal.z),
                reinterpret_cast<unsigned int&>(texcoord.x), reinterpret_cast<unsigned int&>(texcoord.y),
                reinterpret_cast<unsigned int&>(tangent_vector.x), reinterpret_cast<unsigned int&>(tangent_vector.y),
                reinterpret_cast<unsigned int&>(tangent_vector.z));
        } else if (dist_far > ray_tmin && dist_far < ray_tmax) {
            float3 shading_normal = box_normal(dist_far, t0, t1);
            float2 texcoord;
            float3 tangent_vector;
            p = ray_orig + dist_far * ray_dir;

            // calculate uvs and tangent vector
            if (abs(shading_normal.x) > 0.5) {
                texcoord = make_float2((p.y + 0.5), (p.z + 0.5) * shading_normal.x);
                tangent_vector = make_float3(0, 1, 0);
            } else if (abs(shading_normal.y) > 0.5) {
                texcoord = make_float2((p.x + 0.5), -(p.z + 0.5) * shading_normal.y);
                tangent_vector = make_float3(1, 0, 0);
            } else {
                texcoord = make_float2((p.x + 0.5), (p.y + 0.5) * shading_normal.z);
                tangent_vector = make_float3(1, 0, 0);
            }
            optixReportIntersection(
                dist_far, 0, reinterpret_cast<unsigned int&>(shading_normal.x),
                reinterpret_cast<unsigned int&>(shading_normal.y), reinterpret_cast<unsigned int&>(shading_normal.z),
                reinterpret_cast<unsigned int&>(texcoord.x), reinterpret_cast<unsigned int&>(texcoord.y),
                reinterpret_cast<unsigned int&>(tangent_vector.x), reinterpret_cast<unsigned int&>(tangent_vector.y),
                reinterpret_cast<unsigned int&>(tangent_vector.z));
        }
    }
}

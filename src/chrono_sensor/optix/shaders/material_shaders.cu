#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist, Han Wang, Yan Xiao
// =============================================================================
//
// RT kernels for material shading
//
// =============================================================================

#ifdef _WIN32
    #ifndef NOMINMAX
        #define NOMINMAX
    #endif
#endif

#include "chrono_sensor/optix/shaders/device_utils.h"

#ifdef USE_SENSOR_NVDB
#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/Ray.h>
#include <nanovdb/util/HDDA.h>
#endif

static __device__ __inline__ float3 CrossProduct(const float3& a, const float3& b){

    return {(a.y * b.z) - (b.y * a.z),
            (a.z * b.x) - (b.z * a.x),
            (a.x * b.y) - (b.x * a.y)};
}

static __device__ __inline__ float NormalDist(const float& NdH, const float& roughness) {
    float rough_sqr = roughness * roughness;
    float den_2 = NdH * NdH * (rough_sqr - 1.f) + 1.f;
    float denominator = den_2 * den_2;
    return rough_sqr / denominator;
}

// algorithm reference: https://www.gdcvault.com/play/1024478/PBR-Diffuse-Lighting-for-GGX
static __device__ __inline__ float HammonSmith(float NdV, float NdL, const float& roughness) {
    NdV = abs(NdV);
    NdL = abs(NdL);
    float denominator = lerp(2.f * NdV * NdL, NdL + NdV, roughness);
    return 0.5f / denominator;
}

// triangle mesh querie information
__device__ __inline__ void GetTriangleData(float3& normal,
                                           unsigned int& mat_id,
                                           float2& uv,
                                           float3& tangent,
                                           const unsigned int& mesh_id) {
    const int tri_id = optixGetPrimitiveIndex();
    const float2 bary_coord = optixGetTriangleBarycentrics();

    const MeshParameters& mesh_params = params.mesh_pool[mesh_id];
    const uint4& vertex_idx = mesh_params.vertex_index_buffer[tri_id];

    const float3& v1 = make_float3(mesh_params.vertex_buffer[vertex_idx.x]);
    const float3& v2 = make_float3(mesh_params.vertex_buffer[vertex_idx.y]);
    const float3& v3 = make_float3(mesh_params.vertex_buffer[vertex_idx.z]);

    // calculate normales either from normal buffer or vertex positions
    if (mesh_params.normal_index_buffer &&
        mesh_params.normal_buffer) {  // use vertex normals if normal index buffer exists
        const uint4& normal_idx = mesh_params.normal_index_buffer[tri_id];

        normal = normalize(make_float3(mesh_params.normal_buffer[normal_idx.y]) * bary_coord.x +
                           make_float3(mesh_params.normal_buffer[normal_idx.z]) * bary_coord.y +
                           make_float3(mesh_params.normal_buffer[normal_idx.x]) * (1.0f - bary_coord.x - bary_coord.y));

    } else {  // else use face normals calculated from vertices
        normal = normalize(Cross(v2 - v1, v3 - v1));
    }

    // calculate texcoords if they exist
    if (mesh_params.uv_index_buffer && mesh_params.uv_buffer) {  // use vertex normals if normal index buffer exists
        const uint4& uv_idx = mesh_params.uv_index_buffer[tri_id];
        const float2& uv1 = mesh_params.uv_buffer[uv_idx.x];
        const float2& uv2 = mesh_params.uv_buffer[uv_idx.y];
        const float2& uv3 = mesh_params.uv_buffer[uv_idx.z];

        uv = uv2 * bary_coord.x + uv3 * bary_coord.y + uv1 * (1.0f - bary_coord.x - bary_coord.y);
        float3 e1 = v2 - v1;
        float3 e2 = v3 - v1;
        float2 delta_uv1 = uv2 - uv1;
        float2 delta_uv2 = uv3 - uv1;
        float f = 1.f / (delta_uv1.x * delta_uv2.y - delta_uv2.x * delta_uv1.y);
        tangent.x = f * (delta_uv2.y * e1.x - delta_uv1.y * e2.x);
        tangent.y = f * (delta_uv2.y * e1.y - delta_uv1.y * e2.y);
        tangent.z = f * (delta_uv2.y * e1.z - delta_uv1.y * e2.z);
        tangent = normalize(tangent);
    } else {
        uv = make_float2(0.f);
        tangent = make_float3(0.f);
    }

    // get material index
    if (mesh_params.mat_index_buffer) {                  // use vertex normals if normal index buffer exists
        mat_id += mesh_params.mat_index_buffer[tri_id];  // the material index gives an offset id
    }
}

//=============================
// Calculating Refracted color
//=============================

static __device__ __inline__ float3 CalculateRefractedColor(
    PerRayData_camera* prd_camera,
    const int& num_blended_materials,
    unsigned int& material_id,
    const float2& uv,
    const float3& hit_point,
    const float3& ray_dir){

    float accumulated_transparency = 0.f;
    {
        for (int b = 0; b < num_blended_materials; b++) {
            // accumulate transparency by multiplication
            const MaterialParameters& mat = params.material_pool[material_id + b];
            float mat_opacity = mat.transparency;
            if (mat.opacity_tex) {  // override value with a texture if available
                mat_opacity = tex2D<float>(mat.opacity_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
            }
            float mat_blend_weight = 1.f / num_blended_materials;
            if (mat.weight_tex) {  // override blending with weight texture if available
                mat_blend_weight = tex2D<float>(mat.weight_tex, uv.x, uv.y);
            }
            accumulated_transparency += mat_blend_weight * mat_opacity;
        }
    }

    float3 refracted_color = make_float3(0);
    {
        if (accumulated_transparency < 1.f - 1 / 255.f) {
            float3 refract_importance = prd_camera->contrib_to_pixel * (1 - accumulated_transparency);
            if (fmaxf(refract_importance) > params.importance_cutoff && prd_camera->depth + 1 < params.max_depth) {
                PerRayData_camera prd_refraction = default_camera_prd();
                prd_refraction.integrator = prd_camera->integrator;
                prd_refraction.contrib_to_pixel = refract_importance;
                prd_refraction.rng = prd_camera->rng;
                prd_refraction.depth = prd_camera->depth + 1;
                unsigned int opt1, opt2;
                pointer_as_ints(&prd_refraction, opt1, opt2);

                // make_camera_data(make_float3(0), refract_importance, prd_camera.rnd, prd_camera.depth + 1);
                // float3 refract_dir = refract(optixGetWorldRayDirection(), world_normal, 1.f, 1.f);
                float3 refract_dir = ray_dir;  // pure transparency without refraction
                unsigned int raytype = (unsigned int)CAMERA_RAY_TYPE;
                optixTrace(params.root, hit_point, refract_dir, params.scene_epsilon, 1e16f, optixGetRayTime(),
                           OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);
                refracted_color = prd_refraction.color;  // TODO: not sure added here or not
            }
        }
    }

    return refracted_color;
}

//=====================================================
// Calculating Surface reflection toward light sources
//=====================================================

static __device__ __inline__ float3 CalculateReflectedColor(
    PerRayData_camera* prd_camera,
    const int& num_blended_materials,
    unsigned int& material_id,
    const float2& uv,
    const float3& hit_point,
    const float3& world_normal,
    const float3& ray_dir){

    float NdV = Dot(world_normal, -ray_dir);
    float3 light_reflected_color = make_float3(0.0f);
    {
        // iterate through the lights
        for (int i = 0; i < params.num_lights; i++) {
            Light light = params.lights[i];
            if (light.type != LightType::POINT_LIGHT)
                continue;
            PointLight& l = static_cast<PointLight&>(light);//params.lights[i];
            float dist_to_light = Length(l.pos - hit_point);
            if (dist_to_light < 2 * l.max_range) {
                
                float3 dir_to_light = normalize(l.pos - hit_point);
                float NdL = Dot(world_normal, dir_to_light);
                

                // if we think we can see the light, let's see if we are correct
                if (NdL > 0.0f) {
                    // check shadows
                    PerRayData_shadow prd_shadow = default_shadow_prd();
                    prd_shadow.depth = prd_camera->depth + 1;
                    prd_shadow.ramaining_dist = dist_to_light;
                    unsigned int opt1;
                    unsigned int opt2;
                    pointer_as_ints(&prd_shadow, opt1, opt2);
                    unsigned int raytype = (unsigned int)SHADOW_RAY_TYPE;
                    optixTrace(params.root, hit_point, dir_to_light, params.scene_epsilon, dist_to_light,
                               optixGetRayTime(), OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2,
                               raytype);

                    float3 light_attenuation = prd_shadow.attenuation;

                    float point_light_falloff =
                        (l.max_range * l.max_range / (dist_to_light * dist_to_light + l.max_range * l.max_range));

                    float3 incoming_light_ray = l.color * light_attenuation * point_light_falloff * NdL;

                    if (fmaxf(incoming_light_ray) > 0.0f) {
                        float3 halfway = normalize(dir_to_light - ray_dir);
                        float NdV = Dot(world_normal, -ray_dir);
                        float NdH = Dot(world_normal, halfway);
                        float VdH = Dot(-ray_dir, halfway);

                        for (int b = 0; b < num_blended_materials; b++) {
                            const MaterialParameters& mat = params.material_pool[material_id + b];
                            float3 subsurface_albedo = mat.Kd;
                            if (mat.kd_tex) {
                                const float4 tex = tex2D<float4>(mat.kd_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                                // transfer sRGB texture into linear color space.
                                subsurface_albedo = Pow(make_float3(tex.x, tex.y, tex.z), 2.2);
                            }
                            float roughness = mat.roughness;
                            if (mat.roughness_tex) {
                                roughness = tex2D<float>(mat.roughness_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                            }
                            float metallic = mat.metallic;
                            if (mat.metallic_tex) {
                                metallic = tex2D<float>(mat.metallic_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                            }
                            float transparency = mat.transparency;
                            if (mat.opacity_tex) {  // override value with a texture if available
                                transparency = tex2D<float>(mat.opacity_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                            }
                            float mat_blend_weight = 1.f / num_blended_materials;
                            if (mat.weight_tex) {  // override blending with weight texture if available
                                mat_blend_weight = tex2D<float>(mat.weight_tex, uv.x, uv.y);
                            }

                            float3 F = make_float3(0.0f);
                            // float3 subsurface_albedo_updated = subsurface_albedo;
                            // === dielectric workflow
                            if (mat.use_specular_workflow) {
                                float3 specular = mat.Ks;
                                if (mat.ks_tex) {
                                    const float4 tex = tex2D<float4>(mat.ks_tex,uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                                    specular = make_float3(tex.x, tex.y, tex.z);
                                }
                                float3 F0 = specular * 0.08f;
                                F = fresnel_schlick(VdH, 5.f, F0,
                                                    make_float3(1.f) /*make_float3(fresnel_max) it is usually 1*/);
                            } else {
                                float3 default_dielectrics_F0 = make_float3(0.04f);
                                F = metallic * subsurface_albedo + (1 - metallic) * default_dielectrics_F0;
                                subsurface_albedo = subsurface_albedo *
                                                    (1 - metallic);  // since imetals do not do subsurface reflection
                            }

                            // Diffuse portion of reflection
                            float3 contrib_weight =
                                prd_camera->contrib_to_pixel * transparency *
                                mat_blend_weight;  // correct for transparency, light bounces, and blend weight
                            light_reflected_color +=
                                ((make_float3(1.f) - F) * subsurface_albedo * incoming_light_ray) * contrib_weight;
                            float D = NormalDist(NdH, roughness);        // 1/pi omitted
                            float G = HammonSmith(NdV, NdL, roughness);  // 4  * NdV * NdL omitted
                            float3 f_ct = F * D * G;
                            light_reflected_color += f_ct * incoming_light_ray * contrib_weight;
                        }
                    }
                }
            }
        }

        for(int i = 0; i < params.num_lights; i++){
            Light light = params.lights[i];
            if (light.type != LightType::AREA_LIGHT)
                continue;
            AreaLight& a = static_cast<AreaLight&>(light);  

            float dist_to_light = Length(a.pos - hit_point); 
            float3 normal = CrossProduct(a.du, a.dv);
            
            if (dist_to_light < 2 * a.max_range) {
                
                float3 sampleColor = make_float3(0.0f);

                for(int lightSampleID = 0; lightSampleID < 5; lightSampleID++){
                    
                    float3 tempPos = a.pos + (hiprand_uniform(&prd_camera->rng)*a.du)
                                    + (hiprand_uniform(&prd_camera->rng)*a.dv);
                    
                    float3 dir_to_light = normalize(tempPos - hit_point);
                    float NdL = Dot(world_normal, dir_to_light);

                    // Dot product of normal of area light and direction to light
                    // float AdL = Dot(a.normal, dir_to_light);

                    // Checking to see if we can hit light rays towards the source and the orientation of the area light
                    // Allows the light ray to hit light-emitting surface part of area light
                    
                    if (NdL > 0.0f) {
                        // check shadows
                        PerRayData_shadow prd_shadow = default_shadow_prd();
                        prd_shadow.depth = prd_camera->depth + 1;
                        prd_shadow.ramaining_dist = dist_to_light;
                        unsigned int opt1;
                        unsigned int opt2;
                        pointer_as_ints(&prd_shadow, opt1, opt2);
                        unsigned int raytype = (unsigned int)SHADOW_RAY_TYPE;

                        // TODO: Re-implement this multiple times with slightly different dir_to_light values to improve data sampling

                        optixTrace(params.root, hit_point, dir_to_light, params.scene_epsilon, dist_to_light,
                                optixGetRayTime(), OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2,
                                raytype);

                        float3 light_attenuation = prd_shadow.attenuation;

                        float point_light_falloff = (a.max_range * a.max_range / (dist_to_light * dist_to_light + a.max_range * a.max_range));
                        float3 incoming_light_ray = a.color * light_attenuation * point_light_falloff * NdL;

                        if (fmaxf(incoming_light_ray) > 0.0f) {

                            float3 halfway = normalize(dir_to_light - ray_dir);
                            float NdV = Dot(world_normal, -ray_dir);
                            float NdH = Dot(world_normal, halfway);
                            float VdH = Dot(-ray_dir, halfway);

                            for (int b = 0; b < num_blended_materials; b++) {
                                const MaterialParameters& mat = params.material_pool[material_id + b];
                                float3 subsurface_albedo = mat.Kd;
                                
                                if (mat.kd_tex) {
                                    const float4 tex = tex2D<float4>(mat.kd_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                                    // transfer sRGB texture into linear color space.
                                    subsurface_albedo = Pow(make_float3(tex.x, tex.y, tex.z), 2.2);
                                }

                                float roughness = mat.roughness;
                                if (mat.roughness_tex) {
                                    roughness = tex2D<float>(mat.roughness_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                                }
                                float metallic = mat.metallic;
                                if (mat.metallic_tex) {
                                    metallic = tex2D<float>(mat.metallic_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                                }
                                float transparency = mat.transparency;
                                if (mat.opacity_tex) {  // override value with a texture if available
                                    transparency = tex2D<float>(mat.opacity_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                                }
                                float mat_blend_weight = 1.f / num_blended_materials;
                                if (mat.weight_tex) {  // override blending with weight texture if available
                                    mat_blend_weight = tex2D<float>(mat.weight_tex, uv.x, uv.y);
                                }

                                float3 F = make_float3(0.0f);
                                // float3 subsurface_albedo_updated = subsurface_albedo;
                                // === dielectric workflow
                                if (mat.use_specular_workflow) {
                                    float3 specular = mat.Ks;
                                    if (mat.ks_tex) {
                                        const float4 tex = tex2D<float4>(mat.ks_tex,uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                                        specular = make_float3(tex.x, tex.y, tex.z);
                                    }
                                    float3 F0 = specular * 0.08f;
                                    F = fresnel_schlick(VdH, 5.f, F0,
                                                        make_float3(1.f) /*make_float3(fresnel_max) it is usually 1*/);
                                } else {
                                    float3 default_dielectrics_F0 = make_float3(0.04f);
                                    F = metallic * subsurface_albedo + (1 - metallic) * default_dielectrics_F0;
                                    subsurface_albedo = subsurface_albedo *
                                                        (1 - metallic);  // since imetals do not do subsurface reflection
                                }

                                // Diffuse portion of reflection
                                float3 contrib_weight =
                                    prd_camera->contrib_to_pixel * transparency *
                                    mat_blend_weight;  // correct for transparency, light bounces, and blend weight
                                sampleColor +=
                                    ((make_float3(1.f) - F) * subsurface_albedo * incoming_light_ray) * contrib_weight;
                                float D = NormalDist(NdH, roughness);        // 1/pi omitted
                                float G = HammonSmith(NdV, NdL, roughness);  // 4  * NdV * NdL omitted

                                float3 f_ct = F * D * G;
                                sampleColor += f_ct * incoming_light_ray * contrib_weight;
                            }
                        }
                    }
                }

                sampleColor = sampleColor / 5;
                light_reflected_color += sampleColor;
            }
        }
    }

    return light_reflected_color;
}

//===========================
// Calculating Ambient Light
//===========================

static __device__ __inline__ float3 CalculateAmbientLight(
    PerRayData_camera* prd_camera,
    const int& num_blended_materials,
    unsigned int& material_id,
    const float2& uv,
    const float3& world_normal,
    const float3& ray_dir){

    float3 ambient_light = make_float3(0.0f);
    {
        if (!prd_camera->use_gi) {
            float NdV = Dot(world_normal, -ray_dir);
            for (int b = 0; b < num_blended_materials; b++) {
                const MaterialParameters& mat = params.material_pool[material_id + b];
                float3 subsurface_albedo = mat.Kd;
                if (mat.kd_tex) {
                    const float4 tex = tex2D<float4>(mat.kd_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                    // transfer sRGB texture into linear color space.
                    subsurface_albedo = Pow(make_float3(tex.x, tex.y, tex.z), 2.2);
                }
                float transparency = mat.transparency;
                if (mat.opacity_tex) {  // override value with a texture if available
                    transparency = tex2D<float>(mat.opacity_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                }
                float mat_blend_weight = 1.f / num_blended_materials;
                if (mat.weight_tex) {  // override blending with weight texture if available
                    mat_blend_weight = tex2D<float>(mat.weight_tex, uv.x, uv.y);
                }

                float3 contrib_weight = prd_camera->contrib_to_pixel * transparency *
                                       mat_blend_weight;  // correct for transparency, light bounces, and blend weight

                // ambient light model is partial "flashlight" ambient light, partially from normal direction
                ambient_light += params.ambient_light_color *
                                 (make_float3(NdV) + make_float3(Dot(world_normal, make_float3(0, 0, 1)) * .5f + .5f)) *
                                 subsurface_albedo * contrib_weight;
            }
        }
    }

    return ambient_light;
}

//===============================================================
// If the surface is very smoooth, trace the reflected direction
// Do this reflection regardless of GI on or off.
//===============================================================

static __device__ __inline__ float3 CalculateContributionToPixel(
    PerRayData_camera* prd_camera,
    const int& num_blended_materials,
    unsigned int& material_id,
    const float2& uv,
    const float3& world_normal,
    const float3& ray_dir,
    const float3& hit_point){
    
    float NdV = Dot(world_normal, -ray_dir);
    float3 next_contrib_to_pixel = make_float3(0.f);
    float3 next_dir = normalize(reflect(ray_dir, world_normal));
    {
        float NdL = Dot(world_normal, next_dir);
        float3 halfway = normalize(next_dir - ray_dir);
        float NdH = Dot(world_normal, halfway);
        float VdH = Dot(-ray_dir, halfway);  // Same as LdH

        for (int b = 0; b < num_blended_materials; b++) {
            const MaterialParameters& mat = params.material_pool[material_id + b];
            float3 subsurface_albedo = mat.Kd;
            if (mat.kd_tex) {
                const float4 tex = tex2D<float4>(mat.kd_tex, uv.x, uv.y);
                // transfer sRGB texture into linear color space.
                subsurface_albedo = Pow(make_float3(tex.x, tex.y, tex.z), 2.2);
            }
            float roughness = mat.roughness;
            if (mat.roughness_tex) {
                roughness = tex2D<float>(mat.roughness_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
            }
            float metallic = mat.metallic;
            if (mat.metallic_tex) {
                metallic = tex2D<float>(mat.metallic_tex,uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
            }
            float transparency = mat.transparency;
            if (mat.opacity_tex) {  // override value with a texture if available
                transparency = tex2D<float>(mat.opacity_tex,uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
            }
            float mat_blend_weight = 1.f / num_blended_materials;
            if (mat.weight_tex) {  // override blending with weight texture if available
                mat_blend_weight = tex2D<float>(mat.weight_tex, uv.x, uv.y);
            }

            float3 F = make_float3(0.0f);
            // === dielectric workflow
            if (mat.use_specular_workflow) {
                float3 specular = mat.Ks;
                if (mat.ks_tex) {
                    const float4 tex = tex2D<float4>(mat.ks_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                    specular = make_float3(tex.x, tex.y, tex.z);
                }
                float3 F0 = specular * 0.08f;
                F = fresnel_schlick(VdH, 5.f, F0, make_float3(1.f) /*make_float3(fresnel_max) it is usually 1*/);
            } else {
                float3 default_dielectrics_F0 = make_float3(0.04f);
                F = metallic * subsurface_albedo + (1 - metallic) * default_dielectrics_F0;
            }

            float D = NormalDist(NdH, roughness);        // 1/pi omitted
            float G = HammonSmith(NdV, NdL, roughness);  // 4  * NdV * NdL omitted

            float3 f_ct = F * D * G;

            // Note only specular part appears here. Energy preserve
            // Since it is not random, PDF is 1 (normally 1/pi),
            // If the camera uses GI, then it will trace two rays. So each ray's contribution should be halfed

            // corrected for transparency, bounce contribution, and blend
            float weight = transparency * mat_blend_weight;

            // mirror correction accounts for us oversampling this direction
            // following line comes from a heuristic. Perect reflection for metalic smooth objects,
            // no reflection for rough non-metalic objects
            float mirror_correction = (1.f - roughness) * (1.f - roughness) * metallic * metallic;

            // if global illumination, ray contrib will be halved since two rays are propogated
            if (prd_camera->use_gi) {
                weight = weight*.5f;
            }

            float3 partial_contrib = mirror_correction * weight * f_ct * NdL / (4 * HIP_PI_F);
            partial_contrib = clamp(partial_contrib, make_float3(0), make_float3(1));

            partial_contrib = partial_contrib * prd_camera->contrib_to_pixel;

            next_contrib_to_pixel += partial_contrib;
            next_contrib_to_pixel = clamp(next_contrib_to_pixel, make_float3(0), make_float3(1));
        }
    }

    float3 mirror_reflection_color = make_float3(0.0);
    {
        if (luminance(next_contrib_to_pixel) > params.importance_cutoff && prd_camera->depth + 1 < params.max_depth) {
            PerRayData_camera prd_reflection = default_camera_prd();
            prd_reflection.integrator = prd_camera->integrator;
            prd_reflection.contrib_to_pixel = next_contrib_to_pixel;
            prd_reflection.rng = prd_camera->rng;
            prd_reflection.depth = prd_camera->depth + 1;
            prd_reflection.use_gi = prd_camera->use_gi;
            unsigned int opt1, opt2;
            pointer_as_ints(&prd_reflection, opt1, opt2);
            unsigned int raytype = (unsigned int)CAMERA_RAY_TYPE;
            optixTrace(params.root, hit_point, next_dir, params.scene_epsilon, 1e16f, optixGetRayTime(),
                       OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);

            mirror_reflection_color = prd_reflection.color;
        }
    }

    return mirror_reflection_color;
}

//=================
// Global illumination ray
//=================

static __device__ __inline__ float3 CalculateGIReflectionColor(
    PerRayData_camera* prd_camera,
    const int& num_blended_materials,
    unsigned int& material_id,
    const float2& uv,
    const float3& world_normal,
    const float3& ray_dir,
    const float3& hit_point,
    const float3& mirror_reflection_color){

    float NdV = Dot(world_normal, -ray_dir);
    float3 gi_reflection_color = make_float3(0);

    if (prd_camera->use_gi) {
        // sample hemisphere for next ray when using global illumination
        float z1 = hiprand_uniform(&prd_camera->rng);
        float z2 = hiprand_uniform(&prd_camera->rng);
        float3 next_dir = sample_hemisphere_dir(z1, z2, world_normal);

        float NdL = Dot(world_normal, next_dir);
        float3 halfway = normalize(next_dir - ray_dir);

        float NdH = Dot(world_normal, halfway);
        float VdH = Dot(-ray_dir, halfway);  // Same as LdH

        float3 next_contrib_to_pixel = make_float3(0.f);

        for (int b = 0; b < num_blended_materials; b++) {
            const MaterialParameters& mat = params.material_pool[material_id + b];
            float3 subsurface_albedo = mat.Kd;
            if (mat.kd_tex) {
                const float4 tex = tex2D<float4>(mat.kd_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                // transfer sRGB texture into linear color space.
                subsurface_albedo = Pow(make_float3(tex.x, tex.y, tex.z), 2.2);
            }
            float roughness = mat.roughness;
            if (mat.roughness_tex) {
                roughness = tex2D<float>(mat.roughness_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
            }
            float metallic = mat.metallic;
            if (mat.metallic_tex) {
                metallic = tex2D<float>(mat.metallic_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
            }
            float transparency = mat.transparency;
            if (mat.opacity_tex) {  // override value with a texture if available
                transparency = tex2D<float>(mat.opacity_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
            }
            float mat_blend_weight = 1.f / num_blended_materials;
            if (mat.weight_tex) {  // override blending with weight texture if available
                mat_blend_weight = tex2D<float>(mat.weight_tex, uv.x, uv.y);
            }

            float3 F = make_float3(0.0f);
            // === dielectric workflow
            if (mat.use_specular_workflow) {
                float3 specular = mat.Ks;
                if (mat.ks_tex) {
                    const float4 tex = tex2D<float4>(mat.ks_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                    specular = make_float3(tex.x, tex.y, tex.z);
                }
                float3 F0 = specular * 0.08f;
                F = fresnel_schlick(VdH, 5.f, F0, make_float3(1.f) /*make_float3(fresnel_max) it is usually 1*/);
            } else {
                float3 default_dielectrics_F0 = make_float3(0.04f);
                F = metallic * subsurface_albedo + (1 - metallic) * default_dielectrics_F0;
                subsurface_albedo = subsurface_albedo*(1 - metallic);  // since metals do not do subsurface reflection
            }

            float D = NormalDist(NdH, roughness);        // 1/pi omitted
            float G = HammonSmith(NdV, NdL, roughness);  // 4  * NdV * NdL omitted
            float3 f_ct = F * D * G;

            // corrected for transparency, bounce contribution, and blend
            float3 weight = transparency * prd_camera->contrib_to_pixel * mat_blend_weight;

            // If mirror_reflection, then it will trace two rays. So each ray's contribution should be halfed
            if ((mirror_reflection_color.x < 1e-6) && (mirror_reflection_color.y < 1e-6) && (mirror_reflection_color.z < 1e-6)) {
                weight = weight*.5f;
            }

            // Specular part
            next_contrib_to_pixel += weight * f_ct * NdL;

            // Diffuse part
            F = clamp(F, make_float3(0), make_float3(1));
            next_contrib_to_pixel += weight * (make_float3(1.f) - F) * subsurface_albedo * NdL;
        }

        if (luminance(next_contrib_to_pixel) > params.importance_cutoff &&
            prd_camera->depth + 1 < params.max_depth) {
            PerRayData_camera prd_reflection = default_camera_prd();
            prd_reflection.integrator = prd_camera->integrator;
            prd_reflection.contrib_to_pixel = next_contrib_to_pixel;
            prd_reflection.rng = prd_camera->rng;
            prd_reflection.depth = prd_camera->depth + 1;
            prd_reflection.use_gi = prd_camera->use_gi;
            unsigned int opt1, opt2;
            pointer_as_ints(&prd_reflection, opt1, opt2);
            unsigned int raytype = (unsigned int)CAMERA_RAY_TYPE;
            optixTrace(params.root, hit_point, next_dir, params.scene_epsilon, 1e16f, optixGetRayTime(),
                        OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);
            gi_reflection_color = prd_reflection.color;  // accumulate indirect lighting color
        }
    }

    return gi_reflection_color;
}

static __device__ __inline__ void CameraShader(PerRayData_camera* prd_camera,
                                               const MaterialRecordParameters* mat_params,
                                               unsigned int& material_id,
                                               const unsigned int& num_blended_materials,
                                               const float3& world_normal,
                                               const float2& uv,
                                               const float3& tangent,
                                               const float& ray_dist,
                                               const float3& ray_orig,
                                               const float3& ray_dir) {
    //printf("MS| d: %d | contr: (%f,%f,%f)\n", prd_camera->depth, prd_camera->contrib_to_pixel.x,  prd_camera->contrib_to_pixel.y, prd_camera->contrib_to_pixel.z);
    float3 hit_point = ray_orig + ray_dir * ray_dist;

    // if not blended materials, check for transparent cards and short circuit on the transparent texture
    const MaterialParameters& mat = params.material_pool[material_id];
    if (num_blended_materials == 1) {
       

        float transparency = mat.transparency;
        // figure out tranparency
        if (mat.kd_tex) {
            const float4 tex = tex2D<float4>(mat.kd_tex,uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
            if (tex.w < 1e-6)
                transparency = 0.f;  // to handle transparent card textures such as tree leaves
        }

        if (mat.opacity_tex) {
            transparency = tex2D<float>(mat.opacity_tex,uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
        }

        // if this is perfectly transparent, we ignore it and trace the next ray (handles things like tree leaf cards)
        if (transparency < 1e-6) {
            if (prd_camera->depth + 1 < params.max_depth) {
                PerRayData_camera prd_refraction = default_camera_prd();
                prd_refraction.integrator = prd_camera->integrator;
                prd_refraction.contrib_to_pixel = prd_camera->contrib_to_pixel;
                prd_refraction.rng = prd_camera->rng;
                prd_refraction.depth = prd_camera->depth + 1;
                unsigned int opt1, opt2;
                pointer_as_ints(&prd_refraction, opt1, opt2);
                unsigned int raytype = (unsigned int)CAMERA_RAY_TYPE;
                optixTrace(params.root, hit_point, ray_dir, params.scene_epsilon, 1e16f, optixGetRayTime(),
                           OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);
                prd_camera->color = prd_refraction.color;
                //account for fog
                if (prd_camera->use_fog && params.fog_scattering > 0.f) {
                    float blend_alpha = expf(-params.fog_scattering * ray_dist);
                    prd_camera->color = blend_alpha * prd_camera->color + (1 - blend_alpha) * params.fog_color;
                }

                // For GI, harmless without GI
                prd_camera->albedo = prd_refraction.albedo;
                prd_camera->normal = prd_refraction.normal;
            }
            return;
        }
    }

    // for each blended material accumulate transparency, and perform traversal
    float3 refracted_color = CalculateRefractedColor(prd_camera, num_blended_materials, material_id, uv, hit_point, ray_dir);

    // for each light, traverse to light, and calculate each material's shading
    float3 light_reflected_color = CalculateReflectedColor(prd_camera, num_blended_materials, material_id, uv, hit_point, world_normal, ray_dir);

    // for each blended material, calculating total ambient light
    float3 ambient_light = CalculateAmbientLight(prd_camera, num_blended_materials, material_id, uv, world_normal, ray_dir);

    // for each blended material accumulate reflection, and perform traversal    
    float3 next_dir = normalize(reflect(ray_dir, world_normal));
    float3 mirror_reflection_color = CalculateContributionToPixel(prd_camera, num_blended_materials, material_id, uv, world_normal, ray_dir, hit_point);

    // send ray in random direction if global illumination enabled, calculate each materia's shading for a combined shading
    float3 gi_reflection_color = CalculateGIReflectionColor(prd_camera, num_blended_materials, material_id, uv, world_normal, ray_dir, hit_point, mirror_reflection_color);
    
    //=================
    // Combine all traced colors together
    //=================
    prd_camera->color = mirror_reflection_color + light_reflected_color + refracted_color;

    prd_camera->color += prd_camera->use_gi ? gi_reflection_color : ambient_light;

    // Add emissive color
    prd_camera->color += (mat.emissive_power * mat.Ke * abs(Dot(world_normal, -ray_dir)));
    

    // apply fog model
    if (prd_camera->use_fog && params.fog_scattering > 0.f) {
        float blend_alpha = expf(-params.fog_scattering * ray_dist);
        prd_camera->color = blend_alpha * prd_camera->color + (1 - blend_alpha) * params.fog_color;
    }

    //printf("Color: (%.2f,%.2f,%.2f)\n", prd_camera->color.x, prd_camera->color.y, prd_camera->color.z);
    if (prd_camera->depth == 2 && prd_camera->use_gi) {
        float3 accumulated_subsurface_albedo = make_float3(0.f);
        for (int b = 0; b < num_blended_materials; b++) {
            const MaterialParameters& mat = params.material_pool[material_id + b];
            float3 subsurface_albedo = mat.Kd;
            if (mat.kd_tex) {
                const float4 tex = tex2D<float4>(mat.kd_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
                // transfer sRGB texture into linear color space.
                subsurface_albedo = Pow(make_float3(tex.x, tex.y, tex.z), 2.2);
            }
            float mat_blend_weight = 1.f / num_blended_materials;
            if (mat.weight_tex) {  // override blending with weight texture if available
                mat_blend_weight = tex2D<float>(mat.weight_tex, uv.x, uv.y);
            }
            accumulated_subsurface_albedo += subsurface_albedo * mat_blend_weight;
        }
        prd_camera->albedo = accumulated_subsurface_albedo;
        prd_camera->normal = world_normal;
    }
    
}

static __device__ __inline__ void LidarShader(PerRayData_lidar* prd_lidar,
                                              const MaterialParameters& mat,
                                              const float3& world_normal,
                                              const float2& uv,
                                              const float3& tangent,
                                              const float& ray_dist,
                                              const float3& ray_orig,
                                              const float3& ray_dir) {
    prd_lidar->range = ray_dist;
    prd_lidar->intensity = mat.lidar_intensity * abs(Dot(world_normal, -ray_dir));
}

static __device__ __inline__ void RadarShader(PerRayData_radar* prd_radar,
                                              const MaterialParameters& mat,
                                              const float3& world_normal,
                                              const float2& uv,
                                              const float3& tangent,
                                              const float& ray_dist,
                                              const float3& ray_orig,
                                              const float3& ray_dir,
                                              const float3& translational_velocity,
                                              const float3& angular_velocity,
                                              const float& objectId) {
    prd_radar->range = ray_dist;
    prd_radar->rcs = mat.radar_backscatter * abs(Dot(world_normal, -ray_dir));
    float3 hit_point = ray_orig + ray_dir * ray_dist;
    float3 origin = optixTransformPointFromObjectToWorldSpace(make_float3(0, 0, 0));
    float3 r = hit_point - origin;

    prd_radar->velocity = translational_velocity + Cross(angular_velocity, r);
    prd_radar->objectId = objectId;
}

static __device__ __inline__ void ShadowShader(PerRayData_shadow* prd,
                                               const MaterialParameters& mat,
                                               const float3& world_normal,
                                               const float2& uv,
                                               const float3& tangent,
                                               const float& ray_dist,
                                               const float3& ray_orig,
                                               const float3& ray_dir) {

    float transparency = mat.transparency;
    if (mat.kd_tex) {
        const float4 tex = tex2D<float4>(mat.kd_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
        if (tex.w < 1e-6)
            transparency = 0.f;  // to handle transparent card textures such as tree leaves
    }
    if (mat.opacity_tex) {
        transparency = tex2D<float>(mat.opacity_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
    }
    float3 hit_point = ray_orig + ray_dir * ray_dist;
    //printf("Hit Point SH: (%f,%f,%f)\n", hit_point.x, hit_point.y, hit_point.z);
    float atten = 1.f - transparency;  // TODO: figure out the attenuation from the material transparency

    // if the occlusion amount is below the
    prd->attenuation = prd->attenuation * atten;

    if (fmaxf(prd->attenuation) > params.importance_cutoff && prd->depth + 1 < params.max_depth) {
        PerRayData_shadow prd_shadow = default_shadow_prd();
        prd_shadow.attenuation = prd->attenuation;
        prd_shadow.depth = prd->depth + 1;
        prd_shadow.ramaining_dist = prd->ramaining_dist - ray_dist;
        unsigned int opt1, opt2;
        pointer_as_ints(&prd_shadow, opt1, opt2);

        float3 hit_point = ray_orig + ray_dist * ray_dir;
        unsigned int raytype = (unsigned int)SHADOW_RAY_TYPE;
        optixTrace(params.root, hit_point, ray_dir, params.scene_epsilon, prd_shadow.ramaining_dist, optixGetRayTime(),
                   OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);

        prd->attenuation = prd_shadow.attenuation;
    }
}

static __device__ __inline__ void SemanticShader(PerRayData_semantic* prd,
                                                 const MaterialParameters& mat,
                                                 const float3& world_normal,
                                                 const float2& uv,
                                                 const float3& tangent,
                                                 const float& ray_dist,
                                                 const float3& ray_orig,
                                                 const float3& ray_dir) {
    prd->class_id = mat.class_id;
    prd->instance_id = mat.instance_id;
}

static __device__ inline void CameraHapkeShader(PerRayData_camera* prd_camera,
                                               const MaterialRecordParameters* mat_params,
                                               unsigned int& material_id,
                                               const unsigned int& num_blended_materials,
                                               const float3& world_normal,
                                               const float2& uv,
                                               const float3& tangent,
                                               const float& ray_dist,
                                               const float3& ray_orig,
                                               const float3& ray_dir){  

        
        //printf("Distance: %.2f\n", ray_dist);
        //prd_camera->color += make_float3(ray_dist, ray_dist, ray_dist);
        //prd_camera->color += make_float3(1.f,1.f,1.f);
        //return;
        //printf("Hapke Shader!\n");
        // float w = 0.32357f; // average single scattering albedo
        // float b = 0.23955f; // shape controlling parameter for the amplitude of backward and forward scatter of particles
        // float c = 0.30452f; // weighting factor that controls the contribution of backward and forward scatter.
        // float B_s0 = 1.80238f;
        // float h_s = 0.07145f;
        // float B_c0 = 0.0f;
        // float h_c = 1.0f;
        // float phi = 0.3f;
        // //float K = 1.0f;
        // float theta_p = 23.4f*(HIP_PI_F/180);


       const MaterialParameters& mat = params.material_pool[material_id]; // Assume no blended materials for now
       float3 subsarface_albedo = mat.Kd;
       float3 specular = mat.Ks;

       // Get Hapke material parameters
       float w = mat.w;
       float b = mat.b;
       float c = mat.c;
       float B_s0 = mat.B_s0;
       float h_s = mat.h_s;
       float B_c0 = 0.0f;
       float h_c = 1.0f;
       float phi = mat.phi;
       float theta_p = mat.theta_p;

       

       float3 hit_point = ray_orig + ray_dir * ray_dist;

       float cos_e = Dot(world_normal, -ray_dir);

       float3 reflected_color = make_float3(0.0f);
       {
           for (int i = 0; i < params.num_lights; i++) {
               Light light = params.lights[i];
               if (light.type != LightType::POINT_LIGHT)
                   continue;
               PointLight& l = static_cast<PointLight&>(light);
               float dist_to_light = Length(l.pos - hit_point);
               //printf("dist_to_light:%.4f\n", dist_to_light);
               if (1) {//dist_to_light < 2 * l.max_range{ // Sun should have infinity range, so this condition will always be true for ths sun
                   float3 dir_to_light = normalize(l.pos - hit_point);
                   float cos_i = Dot(dir_to_light, world_normal);
                   //printf("cos_i:%.2f",cos_i);
                   if (cos_i > 0) {

                       // Cast a shadow ray to see any attenuation of light
                       PerRayData_shadow prd_shadow = default_shadow_prd();
                       prd_shadow.depth = prd_camera->depth + 1;
                       prd_shadow.ramaining_dist = dist_to_light;
                       unsigned int opt1;
                       unsigned int opt2;
                       pointer_as_ints(&prd_shadow, opt1, opt2);
                       unsigned int raytype = (unsigned int)SHADOW_RAY_TYPE;
                       optixTrace(params.root, hit_point, dir_to_light, params.scene_epsilon, dist_to_light, optixGetRayTime(),
                               OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);

                       float3 light_attenuation = prd_shadow.attenuation;

                       float point_light_falloff  = 1.0f; // ??
                       float3 incoming_light_ray = l.color * cos_i * light_attenuation; // Add attenuation later
                       //printf("incoming_light_ray: (%.2f,%.2f,%.2f)\n", incoming_light_ray.x, incoming_light_ray.y, incoming_light_ray.z);
                       if (fmaxf(incoming_light_ray) > 0.0f) {
                           
                           float cos_g = Dot(dir_to_light, -ray_dir);
                           float sin_i = sqrt(1 - (cos_i*cos_i)); // + sqrt
                           float sin_e = sqrt(1 - (cos_e*cos_e));
                           float sin_g = sqrt(1 - (cos_g*cos_g));

                           float tan_i = sin_i/cos_i;
                           float tan_e = sin_e/cos_e;

                           float cot_i = 1/tan_i;
                           float cot_e = 1/tan_e;
                           float cot_i_sq = cot_i * cot_i;
                           float cot_e_sq = cot_e * cot_e;


                           // Calculate Psi
                           float cos_psi = Dot(normalize(dir_to_light - (cos_i*world_normal)), normalize(-ray_dir - (cos_e*world_normal)));
                           float psi = acos(cos_psi);
                           float psi_half = psi/2;
                           float f_psi = expf(-2 * tan(psi_half));
                           float sin_psi_half = sin(psi_half);
                           float sin_psi_half_sq = sin_psi_half * sin_psi_half;
                           float psi_per_pi = psi/HIP_PI_F; // TODO: Define 1/PI as a constant

                           float tan_theta_p = tan(theta_p);
                           float tan_theta_p_sq = tan_theta_p * tan_theta_p;

                           float cot_theta_p = 1/tan_theta_p;
                           float cot_theta_p_sq = cot_theta_p * cot_theta_p;

                           float E_1_i = expf((-2/HIP_PI_F) * cot_theta_p * cot_i);
                           float E_2_i = expf((-1/HIP_PI_F) * cot_theta_p * cot_theta_p * cot_i * cot_i);

                           float E_1_e = expf((-2/HIP_PI_F) * cot_theta_p * cot_e);
                           float E_2_e = expf((-1/HIP_PI_F) * cot_theta_p * cot_theta_p * cot_e * cot_e);

                           float chi_theta_p = 1 / sqrtf(1 + HIP_PI_F * tan_theta_p_sq);
                           
                           float eta_i = chi_theta_p * (cos_i + sin_i * tan_theta_p * E_2_i / (2 - E_1_i));
                           float eta_e = chi_theta_p * (cos_e + sin_e * tan_theta_p * E_2_e / (2 - E_1_e));

                           float mu0 = cos_i;
                           float mu = cos_e;
                           float mu0_e = chi_theta_p;
                           float mu_e = chi_theta_p;
                           float S = 0.0f;
                           if (cos_i >= cos_g) { // for x,y \in [0,pi], if x <= y => cos(x) >= cos(y)
                               mu0_e *= cos_i + sin_i * tan_theta_p * (cos_psi * E_2_e + sin_psi_half_sq * E_2_i) / (2 - E_1_e - psi_per_pi * E_1_i);

                               mu_e *= cos_e + sin_e * tan_theta_p * (E_2_e - sin_psi_half_sq * E_2_i) / (2 - E_1_e - psi_per_pi * E_1_i);

                               S = mu_e / eta_e * mu0 / eta_i * chi_theta_p / (1 - f_psi + f_psi * chi_theta_p * (mu0/eta_i));
                           }else {
                               mu0_e *= cos_i + sin_i * tan_theta_p * (E_2_i - sin_psi_half_sq * E_2_e) / (2 - E_1_i - psi_per_pi * E_1_e);

                               mu_e *= cos_e + sin_e * tan_theta_p * (cos_psi * E_2_i + sin_psi_half_sq * E_2_e) / (2 - E_1_i - psi_per_pi * E_1_e);

                               S = mu_e / eta_e * mu0 / eta_i * chi_theta_p / (1 - f_psi + f_psi * chi_theta_p * (mu/eta_e));
                           }

                           float KPhi = 1.209 * pow(phi, 2.0f/3);
                           float K = -log(1 - KPhi)/KPhi;

                           float tan_ghalf = sin_g / (1 + cos_g);
                           float tan_ghalf_per_hC = tan_ghalf/h_c;

                           float B_C = 0.0f;
                           if (cos_g < 1.0f)
                               B_C = (1 + (1 - exp(-tan_ghalf_per_hC)) / tan_ghalf_per_hC) / (2 * pow(1 + tan_ghalf_per_hC, 2));
                           else if (cos_g == 1)
                               B_C = 1;
                           
                           float r0Term = sqrt(1 - w);
                           float r0 = (1 - r0Term)/(1 + r0Term);

                           float LS = mu0_e / (mu0_e + mu_e);
                           float b_sq = b*b;

                           float twobcos_g = 2 * b * cos_g;

                           float p_g = (1 + c) / 2 * (1-b_sq) / pow(1 - (2*b*cos_g) + b_sq, 1.5f) + (1 - c)/2 * (1-b_sq)/pow(1 + (2*b*cos_g) + b_sq, 1.5f);

                           float B_S = 1 / (1 + tan_ghalf / h_s);

                           float x_i = mu0_e/K;
                           float x_e = mu_e/K;
                           float H_i = 1/(1 - w * x_i * (r0 + (1 - 2 * r0 * x_i) / 2 * log((1+x_i)/x_i)));
                           float H_e = 1/(1 - w * x_e * (r0 + (1 - 2 * r0 * x_e) / 2 * log((1+x_e)/x_e)));

                           float M = H_i * H_e - 1;
                           float f_ct = LS * K * w/(4*HIP_PI_F) * (p_g * (1 + B_s0 * B_S) + M) * (1 + B_c0 * B_C) * S /cos_i;
                           //printf("fct:%.2f\n", f_ct);
                           reflected_color += f_ct * incoming_light_ray * subsarface_albedo;
                           //printf("reflected_color:(%.2f,%.2f,%.2f)\n", reflected_color.x, reflected_color.y, reflected_color.z);
                       }

                   }       
               }
           }
       }
     //printf("reflected_color:(%.2f,%.2f,%.2f)\n", reflected_color.x, reflected_color.y, reflected_color.z);
     prd_camera->color += reflected_color;
}

static __device__ __inline__ float SchlickPhase(float VdL, float k) {
    float numerator = 1 - (k * k);
    float denominator = 4 * HIP_PI * (1 - k * VdL) * (1 - k * VdL);
    return numerator / denominator;
}


static __device__ inline void CameraVolumetricShader(PerRayData_camera* prd_camera,
                                                     const MaterialRecordParameters* mat_params,
                                                     unsigned int& material_id,
                                                     const unsigned int& num_blended_materials,
                                                     const float3& world_normal,
                                                     const float2& uv,
                                                     const float3& tangent,
                                                     const float& ray_dist,
                                                     const float3& ray_orig,
                                                     const float3& ray_dir) {
#ifdef USE_SENSOR_NVDB
    nanovdb::NanoGrid<float>* grid = params.handle_ptr;
    const nanovdb::Vec3f ray_orig_v(ray_orig.x, ray_orig.y, ray_orig.z);
    const nanovdb::Vec3f ray_dir_v(ray_dir.x, ray_dir.y, ray_dir.z);
    float3 hitPoint = ray_orig + ray_dir * ray_dist;

    nanovdb::Vec3d hitPointIdx = grid->worldToIndex(nanovdb::Vec3d(hitPoint.x, hitPoint.y, hitPoint.z));
    nanovdb::Vec3d rayDirIdx = grid->worldToIndex(ray_dir_v);
    nanovdb::Vec3d rayOrigIdx = grid->worldToIndex(ray_orig_v);

    nanovdb::Ray<float> ray(rayOrigIdx, rayDirIdx, ray_dist, 1e20);
    /*   printf("VolShader: ray_dist: %f| hitP: %f,%f,%f | hitP Idx: %f,%f,%f | rayStartIdx: %f %f %f | rayDirIdx:
       %f,%f,%f\n", ray_dist, hitPoint.x, hitPoint.y, hitPoint.z, hitPointIdx[0], hitPointIdx[1], hitPointIdx[2],
       ray.start()[0], ray.start()[1], ray.start()[2], rayDirIdx[0], rayDirIdx[1], rayDirIdx[2]);*/

    nanovdb::Coord ijk = nanovdb::RoundDown<nanovdb::Coord>(ray.start());  // first hit of bbox
    // printf("ZCrossing::ray.start(): (%f,%f,%f) | ray.dir(): (%f,%f,%f)\n", ray.start()[0], ray.start()[1],
    // ray.start()[2], ray.dir()[0],ray.dir()[1],ray.dir()[2]);

    float v;
    nanovdb::DefaultReadAccessor<float> acc = grid->tree().getAccessor();
    nanovdb::HDDA<nanovdb::Ray<float>, nanovdb::Coord> hdda(ray, acc.getDim(ijk, ray));
    const auto v0 = acc.getValue(ijk);

    // printf("Start Value: %f | Start Idx: %f,%f,%f\n", v0, ijk.asVec3d()[0], ijk.asVec3d()[1], ijk.asVec3d()[2]);
    static const float Delta = 1.0001f;
    int nsteps = 0;
    float transmittance = 1.0f;
    float absorptionCoeff = 0.001;
    float scatteringCoeff = 0.01;
    float extinctionCoeff = absorptionCoeff + scatteringCoeff;
    float3 inScattering = make_float3(0);
    float outScattering = 0;
    float k = 0;  // isotropic reflections
    float3 volAlbedo = make_float3(0.659, 0.459, 0.051);

    int inactiveSteps = 0;
    float3 volumeLight = make_float3(0);
    while (hdda.step() && nsteps < 100) {
        ijk = nanovdb::RoundDown<nanovdb::Coord>(ray(hdda.time() + Delta));
        hdda.update(ray, acc.getDim(ijk, ray));
        if (hdda.dim() > 1 || !acc.isActive(ijk)) {
            inactiveSteps++;
            if (inactiveSteps > 1000)
                break;
            continue;  // either a tile value or an inactive voxel
        }

        // sample lights
        while (hdda.step() && acc.isActive(hdda.voxel())) {  // in the narrow band
            v = acc.getValue(hdda.voxel());                  // density
            ijk = hdda.voxel();
            nanovdb::Vec3f volPntIdx =
                grid->indexToWorld(ijk.asVec3s());  // TODO: Make VDB to chrono data type conversion function
            float3 volPnt = make_float3(volPntIdx[0], volPntIdx[1], volPntIdx[2]);
            // printf("density: %f\n", v);
            transmittance *= exp((-v * extinctionCoeff * Delta));
            outScattering = scatteringCoeff * v;

            for (int i = 0; i < params.num_lights; i++) {
                Light l = params.lights[i];
                if (l.type != LightType::POINT_LIGHT)
                    continue;
                float dist_to_light = Length(l.pos - volPnt);
                if (dist_to_light < 2 * l.max_range) {
                    float3 dir_to_light = normalize(l.pos - volPnt);
                    float VdL = Dot(-1 * normalize(ray_dir), -1 * dir_to_light);
                    if (VdL > 0) {
                        float3 light_attenuation = make_float3(0);  // TODO: shoot shadow ray
                        {
                            // Ray march to determine light attenuation
                            nanovdb::Ray<float> sRay(
                                ijk.asVec3d(),
                                grid->worldToIndex(nanovdb::Vec3d(dir_to_light.x, dir_to_light.y, dir_to_light.z)), 0.f,
                                1e20);
                            nanovdb::Coord sijk = nanovdb::RoundDown<nanovdb::Coord>(sRay.start());
                            nanovdb::HDDA<nanovdb::Ray<float>, nanovdb::Coord> shdda(sRay, acc.getDim(sijk, sRay));
                            int sinactiveSteps = 0;
                            int ssteps = 0;
                            float sV = 0;
                            float sTransmittance = 1.0f;
                            while (shdda.step() && ssteps < 50) {
                                sijk = nanovdb::RoundDown<nanovdb::Coord>(sRay(shdda.time() + Delta));
                                shdda.update(sRay, acc.getDim(sijk, sRay));
                                if (shdda.dim() > 1 || !acc.isActive(sijk)) {
                                    sinactiveSteps++;
                                    if (sinactiveSteps > 20)
                                        break;
                                    continue;  // either a tile value or an inactive voxel
                                }
                                while (shdda.step() && acc.isActive(shdda.voxel())) {  // in the narrow band
                                    sV = acc.getValue(shdda.voxel());
                                    sTransmittance *= exp((-sV * extinctionCoeff * Delta));  // density
                                    ssteps++;
                                }
                            }
                            light_attenuation = make_float3(clamp(sTransmittance, 0.f, 1.f));
                        }
                        // printf("Light Atten: %f\n", light_attenuation.x);
                        float point_light_falloff =
                            (l.max_range * l.max_range / (dist_to_light * dist_to_light + l.max_range * l.max_range));

                        float3 incoming_light_ray = l.color * light_attenuation * point_light_falloff * VdL;
                        float phase = SchlickPhase(VdL, k);
                        inScattering = params.ambient_light_color + incoming_light_ray * phase;

                        volumeLight += transmittance * inScattering * outScattering * Delta;
                    }
                }
            }
            nsteps++;
            // break;
        }
    }
    float alpha = 1 - clamp(transmittance, 0, 1);
    if (nsteps > 0) {
        prd_camera->transparency = 1 - alpha;
        prd_camera->color += volumeLight;  // make_float3(1-alpha, 1-alpha, 1-alpha);
        // prd_camera->color += make_float3(0, 0, 1);
    } else {
        // prd_camera->transparency = 1.f;
        prd_camera->color += make_float3(0, 0, 0);  // 0.1f, 0.2f, 0.4f
    }
#endif
}

static __device__ __inline__ void DepthShader(PerRayData_depthCamera* prd,
                                              const MaterialParameters& mat,
                                              const float3& world_normal,
                                              const float2& uv,
                                              const float3& tangent,
                                              const float& ray_dist,
                                              const float3& ray_orig,
                                              const float3& ray_dir) {
    prd->depth = fminf(prd->max_depth, ray_dist);
}

static __device__ inline void SamplePointLight(Light pl, LightSample* ls) {
    ls->dir = normalize(pl.pos - ls->hitpoint); // How much slow down due to derefing hitpoint twice?
    float dist = Length(pl.pos - ls->hitpoint);
    ls->dist = dist;
    ls->pdf = 1.f;
    float point_light_falloff = (pl.max_range * pl.max_range / (dist * dist + pl.max_range * pl.max_range));
    ls->L = pl.color * point_light_falloff;
}

static __device__ inline void SampleSpotLight(Light spot, LightSample* ls) {
    ls->dir = normalize(spot.pos - ls->hitpoint);  // How much slow down due to derefing hitpoint twice?
    float dist = Length(spot.pos - ls->hitpoint);
    ls->dist = dist;
    ls->pdf = 1.f;

    float cos_theta = Dot(spot.spot_dir, -1*ls->dir);
    
    // Replace max range with a high intensity
    //float point_light_falloff = (spot.max_range * spot.max_range / (dist * dist + spot.max_range * spot.max_range));
    ls->L = spot.color / (dist*dist);

    float falloff;  // spot light falloff
    if (cos_theta >= spot.cos_falloff_start) {
        falloff = 1.f;
        return;
    } 
    if (cos_theta < spot.cos_total_width) {
        falloff = 0.f;
        ls->L = make_float3(0.f);
        return;
    }
        
    float delta = (cos_theta - spot.cos_total_width) / (spot.cos_falloff_start - spot.cos_total_width);
    falloff = (delta * delta) * (delta * delta);
   
    ls->L = ls->L * falloff;
    //printf("falloff: %f | dist: %f | cosTheta: %f\n", falloff, dist, cos_theta*180/HIP_PI);
}

static __device__ inline void SampleLight(Light light, LightSample* ls) {
    switch (light.type) {
        case LightType::POINT_LIGHT:
            SamplePointLight(light, ls);
            break;
        case LightType::SPOT_LIGHT:
            //printf("Sample Spot!\n");
            SampleSpotLight(light,ls);
            break;
        default:
            break;
    }
}






static __device__ __inline__ float LambertianBSDFPdf(float3& wo, float3& wi, float3& n) {
    // float WodWi = Dot(wo,wi);
    float NdWi = Dot(n, wi);
    return NdWi > 0 ? NdWi * INV_PI : 0;
}

static __device__ __inline__ void LambertianBSDFSample(BSDFSample& sample, const MaterialParameters& mat, bool eval, float z1, float z2) {
    sample.f = mat.Kd * INV_PI;
    if (eval) return;

    sample.wi = sample_hemisphere_dir(z1, z2, sample.n);
    sample.pdf = LambertianBSDFPdf(sample.wo, sample.wi, sample.n);
}


static __device__ __inline__ BSDFSample SampleBSDF(BSDFType type,
                                                   BSDFSample& sample,
                                                   const MaterialParameters& mat,
                                                   bool eval = false,
                                                   float z1 = 0,
                                                   float z2 = 0) {
    switch (type) {
        case BSDFType::DIFFUSE:
            LambertianBSDFSample(sample, mat, eval, z1, z2);
            break;
        case BSDFType::SPECULAR:
            break;
        case BSDFType::DIELECTRIC:
            break;
        case BSDFType::GLOSSY:
            break;
        case BSDFType::DISNEY:
            break;
        case BSDFType::HAPKE:
            break;
        default:
            break;
    }

    return sample;
}

static __device__ __inline__ float EvalBSDFPDF(BSDFType type, float3& wo, float3& wi, float3& n) {
    float pdf;
    switch (type) {
        case BSDFType::DIFFUSE:
            pdf = LambertianBSDFPdf(wo,wi,n);
            break;
        case BSDFType::SPECULAR:
            break;
        case BSDFType::DIELECTRIC:
            break;
        case BSDFType::GLOSSY:
            break;
        case BSDFType::DISNEY:
            break;
        case BSDFType::HAPKE:
            break;
        default:
            break;
    }

    return pdf;
}

// Importance Sampling power heurustic method
static __device__ __inline__ float ISPowerHeuristic(int nf, float fPdf, int ng, float gPdf) {
    float f = nf * fPdf, g = ng * gPdf;
    return (f * f) / (f * f + g * g);
}

static __device__ __inline__ float3 ComputeDirectLight(Light& l, LightSample& ls, const MaterialParameters& mat, int depth) {
    float3 Ld = make_float3(0.f);
    SampleLight(l, &ls);
    BSDFType bsdfType = (BSDFType)mat.BSDFType;
    if (ls.pdf > 0 && fmaxf(ls.L) > 0) {
        float NdL = Dot(ls.n, ls.dir);
        if (NdL > 0) {
            // Evaluate BSDF at light direction
            BSDFSample bsdf;
            bsdf.wi = ls.dir;
            bsdf.wo = ls.wo;
            bsdf.n = ls.n;
            SampleBSDF(bsdfType, bsdf, mat, true);
            float scatterPDF = EvalBSDFPDF(bsdfType, bsdf.wo, bsdf.wi, bsdf.n);
            if (!(fmaxf(bsdf.f) > 0)) return Ld; // If the BSDF is black, direct light contribution  is 0?
            // Shoot shadow rays
            PerRayData_shadow prd_shadow = default_shadow_prd();
            prd_shadow.depth = depth + 1;
            prd_shadow.ramaining_dist = ls.dist;
            unsigned int opt1;
            unsigned int opt2;
            pointer_as_ints(&prd_shadow, opt1, opt2);
            unsigned int raytype = (unsigned int)SHADOW_RAY_TYPE;
            optixTrace(params.root, ls.hitpoint, ls.dir, params.scene_epsilon, ls.dist, optixGetRayTime(),
                       OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);

            // light contribution
            float3 light_contrib = bsdf.f * NdL * (prd_shadow.attenuation);
           // printf("L Contr: (%f,%f,%f)\n", light_contrib.x,light_contrib.y, light_contrib.z);
            if (l.delta) {
                Ld += light_contrib * ls.L / ls.pdf;
            } else {
                float is_weight = ISPowerHeuristic(1, ls.pdf, 1, scatterPDF);
                Ld += light_contrib * ls.L * is_weight / ls.pdf;
            }
        }
    }

    // TODO:: Add MIS computation for non delta lights

    return Ld;
}

static __device__ __inline__ void CameraPathIntegrator(PerRayData_camera* prd_camera,
                                                        const MaterialRecordParameters* mat_params,
                                                        unsigned int& material_id,
                                                        const unsigned int& num_blended_materials,
                                                        const float3& world_normal,
                                                        const float2& uv,
                                                        const float3& tangent,
                                                        const float& ray_dist,
                                                        const float3& ray_orig,
                                                        const float3& ray_dir) {

    //if (prd_camera->depth >= 3)
    //    printf("PI| d: %d | contr: (%f,%f,%f)\n", prd_camera->depth, prd_camera->contrib_to_pixel.x,
    //           prd_camera->contrib_to_pixel.y, prd_camera->contrib_to_pixel.z);
    const MaterialParameters& mat = params.material_pool[material_id];
    BSDFType bsdfType = (BSDFType)mat.BSDFType;
    float3 hit_point = ray_orig + ray_dir * ray_dist;
    float3 L = make_float3(0.0f);

    float3 wo = -ray_dir;

    // Add ambient light
    //prd_camera->color += params.ambient_light_color * prd_camera->contrib_to_pixel;  // ?

    float3 Le = make_float3(0.f);
    // TODO: Add Emisions from Area Lights

    // Direct light contributions
    float3 Ld = make_float3(0.f);
   
    if (params.num_lights > 0 && bsdfType != BSDFType::SPECULAR) {
        //printf("Direct Light| BSDF: %d != %d\n", bsdfType, BSDFType::SPECULAR);
        // Uniform sample light
        unsigned int sample_light_index = (unsigned int)(hiprand_uniform(&prd_camera->rng) *params.num_lights);  // TODO: Won't work for whitted as no GI, have a global sampler instead?
        Light l = params.lights[sample_light_index];
        LightSample ls;
        ls.hitpoint = hit_point;
        ls.wo = wo;
        ls.n = world_normal;
        
        // Compute direct lighting
        float3 ld = ComputeDirectLight(l, ls, mat, prd_camera->depth);
        //printf("d: %d | DL: (%f,%f,%f) \n", prd_camera->depth, ld.x,ld.y,ld.z);
        Ld = prd_camera->contrib_to_pixel * ComputeDirectLight(l,ls,mat,prd_camera->depth);
    }
    L += Ld;
    
    if (prd_camera->depth + 1 < params.max_depth) {
       // printf("Next ray!\n");
        BSDFSample sample;
        sample.wo = wo;
        sample.n = world_normal;
        float z1 = hiprand_uniform(&prd_camera->rng);
        float z2 = hiprand_uniform(&prd_camera->rng);
        SampleBSDF(bsdfType, sample, mat, false, z1, z2);
        float NdL = Dot(sample.n, sample.wi);
        float3 next_contrib_to_pixel = prd_camera->contrib_to_pixel * sample.f * NdL / sample.pdf; 
        if (luminance(sample.f) > params.importance_cutoff && sample.pdf > 0 && fmaxf(next_contrib_to_pixel) > 0) {         
            // Check possible rr termination
            float rr_thresh = .1f; // Replace this with importance_cutoff?
            if (fmaxf(next_contrib_to_pixel) < rr_thresh && prd_camera->depth > 3) {
                float q = fmaxf((float).05, 1 - fmaxf(next_contrib_to_pixel));
                float p = hiprand_uniform(&prd_camera->rng);
                if (p < q) return;
                next_contrib_to_pixel = next_contrib_to_pixel / (1-q);
            }

            // Trace next ray
            PerRayData_camera prd_reflection = default_camera_prd();
            prd_reflection.integrator = prd_camera->integrator;
            prd_reflection.contrib_to_pixel = next_contrib_to_pixel;
            prd_reflection.rng = prd_camera->rng;
            prd_reflection.depth = prd_camera->depth + 1;
            prd_reflection.use_gi = prd_camera->use_gi;
            unsigned int opt1, opt2;
            pointer_as_ints(&prd_reflection, opt1, opt2);
            unsigned int raytype = (unsigned int)CAMERA_RAY_TYPE;
            optixTrace(params.root, hit_point, sample.wi, params.scene_epsilon, 1e16f, optixGetRayTime(),
                       OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);
            L += prd_reflection.color;
        }
    }

    prd_camera->color += L;
    prd_camera->albedo = mat.Kd; // Might change
    prd_camera->normal = world_normal;
}

static __device__ __inline__ void TransientPathIntegrator(PerRayData_transientCamera* prd_camera,
                                                          const MaterialRecordParameters* mat_params,
                                                          unsigned int& material_id,
                                                          const unsigned int& num_blended_materials,
                                                          const float3& world_normal,
                                                          const float2& uv,
                                                          const float3& tangent,
                                                          const float& ray_dist,
                                                          const float3& ray_orig,
                                                          const float3& ray_dir) {

    //printf("TRANSIENT Integrator!\n");
    const MaterialParameters& mat = params.material_pool[material_id];
    BSDFType bsdfType = (BSDFType)mat.BSDFType;
    float3 hit_point = ray_orig + ray_dir * ray_dist;
    float3 L = make_float3(0.0f);

    float3 wo = -ray_dir;

    prd_camera->path_length += ray_dist;

    // Add ambient light
    // prd_camera->color += params.ambient_light_color * prd_camera->contrib_to_pixel;  // ?

    float3 Le = make_float3(0.f);
    // TODO: Add Emisions from Area Lights

    // Direct light contributions
    float3 Ld = make_float3(0.f);

    if (params.num_lights > 0 && bsdfType != BSDFType::SPECULAR) {
        // Uniform sample light
        unsigned int sample_light_index =
            (unsigned int)(hiprand_uniform(&prd_camera->rng) *
                           params.num_lights);  // TODO: Won't work for whitted as no GI, have a global sampler instead?
        Light l = params.lights[sample_light_index];
        LightSample ls;
        ls.hitpoint = hit_point;
        ls.wo = wo;
        ls.n = world_normal;

        // Compute direct lighting
        Ld = prd_camera->contrib_to_pixel * ComputeDirectLight(l, ls, mat, prd_camera->depth);
  
        if (fmaxf(Ld) > 0) {
        
            TransientSample sample = {};
            sample.pathlength = prd_camera->path_length + ls.dist;
            sample.color = Ld;
            int idx = params.max_depth * prd_camera->current_pixel + (prd_camera->depth - 1);
           
            params.transient_buffer[idx] = sample;
        }
    }
    L += Ld;

    if (prd_camera->depth + 1 < params.max_depth) {
        BSDFSample sample;
        sample.wo = wo;
        sample.n = world_normal;
        float z1 = hiprand_uniform(&prd_camera->rng);
        float z2 = hiprand_uniform(&prd_camera->rng);
        SampleBSDF(bsdfType, sample, mat, false, z1, z2);
        if (luminance(sample.f) > params.importance_cutoff > 0 && sample.pdf > 0) {
            float NdL = Dot(sample.n, sample.wi);
            float3 next_contrib_to_pixel = prd_camera->contrib_to_pixel * sample.f * NdL / sample.pdf;
            if (fmaxf(next_contrib_to_pixel) < 0) {
                prd_camera->depth_reached = prd_camera->depth;
                return;
            }
            // Check possible rr termination
            float rr_thresh = .1f;
            if (fmaxf(next_contrib_to_pixel) < rr_thresh && prd_camera->depth > 3) {
                float q = fmaxf((float).05, 1 - fmaxf(next_contrib_to_pixel));
                float p = hiprand_uniform(&prd_camera->rng);
                if (p < q)
                    return;
                next_contrib_to_pixel = next_contrib_to_pixel / (1 - q);
            }

            // Trace next ray
            PerRayData_transientCamera prd_reflection = default_transientCamera_prd(prd_camera->current_pixel);
            prd_reflection.integrator = prd_camera->integrator;
            prd_reflection.contrib_to_pixel = next_contrib_to_pixel;
            prd_reflection.rng = prd_camera->rng;
            prd_reflection.depth = prd_camera->depth + 1;
            prd_reflection.use_gi = prd_camera->use_gi;
            prd_reflection.path_length = prd_camera->path_length;
            unsigned int opt1, opt2;
            pointer_as_ints(&prd_reflection, opt1, opt2);
            unsigned int raytype = (unsigned int)TRANSIENT_RAY_TYPE;
            optixTrace(params.root, hit_point, sample.wi, params.scene_epsilon, 1e16f, optixGetRayTime(),
                       OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);
            L += prd_reflection.color;
            prd_camera->depth_reached = prd_reflection.depth_reached;
        }
    } else {
        prd_camera->depth_reached = prd_camera->depth;
    }

    prd_camera->color += L;
    prd_camera->albedo = mat.Kd;  // Might change
    prd_camera->normal = world_normal;
}

static __device__ __inline__ float ComputePathLengthImportance(float path_length) {
    float importance = 1.f;
    float v = (path_length - params.target_dist) / params.window_size;
    switch (params.timegated_mode) {
        case TIMEGATED_MODE::BOX: {
            importance = fabsf(v) < .5f ? 1.f : 0.f;
            break;
        } case TIMEGATED_MODE::TENT :{
            importance = fmaxf(1 - fabsf(v),0.f);
            break;
        }
        case TIMEGATED_MODE::COS :{
            importance = cosf(2*HIP_PI*v);
            break;
        }
        case TIMEGATED_MODE::SIN: {
            importance = sinf(2 * HIP_PI * v);
            break;
        }
        case TIMEGATED_MODE::EXPONENTIAL: {
            importance = path_length > params.target_dist ? 0.f : expf(v);
            break;
        }
    }
    return importance;
}

static __device__ __inline__ void TimeGatedIntegrator(PerRayData_transientCamera* prd_camera,
                                                          const MaterialRecordParameters* mat_params,
                                                          unsigned int& material_id,
                                                          const unsigned int& num_blended_materials,
                                                          const float3& world_normal,
                                                          const float2& uv,
                                                          const float3& tangent,
                                                          const float& ray_dist,
                                                          const float3& ray_orig,
                                                          const float3& ray_dir) {
    //printf("TIMEGATED Integrator!\n");
    const MaterialParameters& mat = params.material_pool[material_id];
    BSDFType bsdfType = (BSDFType)mat.BSDFType;
    float3 hit_point = ray_orig + ray_dir * ray_dist;
    float3 L = make_float3(0.0f);

    float3 wo = -ray_dir;

    prd_camera->path_length += ray_dist;

    // Add ambient light
    // prd_camera->color += params.ambient_light_color * prd_camera->contrib_to_pixel;  // ?

    float3 Le = make_float3(0.f);
    // TODO: Add Emisions from Area Lights

    // Direct light contributions
    float3 Ld = make_float3(0.f);

    if (params.num_lights > 0 && bsdfType != BSDFType::SPECULAR) {
        // Uniform sample light
        unsigned int sample_light_index =
            (unsigned int)(hiprand_uniform(&prd_camera->rng) *
                           params.num_lights);  // TODO: Won't work for whitted as no GI, have a global sampler instead?
        Light l = params.lights[sample_light_index];
        LightSample ls;
        ls.hitpoint = hit_point;
        ls.wo = wo;
        ls.n = world_normal;

        // Compute direct lighting
        Ld = prd_camera->contrib_to_pixel * ComputeDirectLight(l, ls, mat, prd_camera->depth);

        if (fmaxf(Ld) > 0) {
            float path_importance = ComputePathLengthImportance(prd_camera->path_length + ls.dist);
            L += (Ld * path_importance);
        }
    }


    if (prd_camera->depth + 1 < params.max_depth) {
        BSDFSample sample;
        sample.wo = wo;
        sample.n = world_normal;
        float z1 = hiprand_uniform(&prd_camera->rng);
        float z2 = hiprand_uniform(&prd_camera->rng);
        SampleBSDF(bsdfType, sample, mat, false, z1, z2);
        if (luminance(sample.f) > params.importance_cutoff > 0 && sample.pdf > 0) {

            float NdL = Dot(sample.n, sample.wi);
            float3 next_contrib_to_pixel = prd_camera->contrib_to_pixel * sample.f * NdL / sample.pdf;
            if (fmaxf(next_contrib_to_pixel) < 0) {
                prd_camera->depth_reached = prd_camera->depth;
                return;
            }
            // Check possible rr termination
            float rr_thresh = .1f;
            if (fmaxf(next_contrib_to_pixel) < rr_thresh && prd_camera->depth > 3) {
                float q = fmaxf((float).05, 1 - fmaxf(next_contrib_to_pixel));
                float p = hiprand_uniform(&prd_camera->rng);
                if (p < q)
                    return;
                next_contrib_to_pixel = next_contrib_to_pixel / (1 - q);
            }

            // Trace next ray
            PerRayData_transientCamera prd_reflection = default_transientCamera_prd(prd_camera->current_pixel);
            prd_reflection.integrator = prd_camera->integrator;
            prd_reflection.contrib_to_pixel = next_contrib_to_pixel;
            prd_reflection.rng = prd_camera->rng;
            prd_reflection.depth = prd_camera->depth + 1;
            prd_reflection.use_gi = prd_camera->use_gi;
            prd_reflection.path_length = prd_camera->path_length;
            unsigned int opt1, opt2;
            pointer_as_ints(&prd_reflection, opt1, opt2);
            unsigned int raytype = (unsigned int)TRANSIENT_RAY_TYPE;
            optixTrace(params.root, hit_point, sample.wi, params.scene_epsilon, 1e16f, optixGetRayTime(),
                       OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);
            L += prd_reflection.color;
            prd_camera->depth_reached = prd_reflection.depth_reached;
        }
    } else {
        prd_camera->depth_reached = prd_camera->depth;
    }

    prd_camera->color += L;
    prd_camera->albedo = mat.Kd;  // Might change
    prd_camera->normal = world_normal;
}

static __device__  __inline__ int binary_search_cdf(const float* cdf, int num_elements, float value) {
    int low = 0;
    int high = num_elements - 1;

    while (low < high) {
        int mid = (low + high) / 2;
        if (cdf[mid] < value) {
            low = mid + 1;
        } else {
            high = mid;
        }
    }
    return low;
}

static __device__ __inline__ void sample_reuse_kernel(const float* cdf,
                                    const float* pmf,
                                    int num_elements,
                                    float* random_samples,
                                    int* sampled_indices,
                                    float* rescaled_samples) {

    //float value = random_samples[idx];

    //// Find the index corresponding to the CDF
    //int index = binary_search_cdf(cdf, num_elements, value);

    //// Compute the PMF and CDF values at the index
    //float pmf_value = pmf[index];
    //float cdf_value = (index > 0) ? cdf[index - 1] : 0.0f;

    //// Rescale the sample for reuse
    //float rescaled_value = (value - cdf_value) / pmf_value;

    //// Store the results
    //sampled_indices[idx] = index;
    //rescaled_samples[idx] = rescaled_value;
}

static __device__ __inline__ float3 EmitterLaserSample() {

}

static __device__ __inline__ void SampleHiddenGeometryPos() {

}

static __device__ __inline__ void HiddenGeometrySample() {

}

static __device__ __inline__ void LaserNEEE(PerRayData_laserSampleRay* prd,
                                            const MaterialRecordParameters* mat_params,
                                            unsigned int& material_id,
                                            const unsigned int& num_blended_materials,
                                            const float3& world_normal,
                                            const float2& uv,
                                            const float3& tangent,
                                            const float& ray_dist,
                                            const float3& ray_orig,
                                            const float3& ray_dir) {

    const MaterialParameters& mat = params.material_pool[material_id];
    BSDFType bsdfType = (BSDFType)mat.BSDFType;
    // Set the hit point manually as the laser focus point to account for floating point errors in optix
    // Since the original hitpoint with error is somehere around the focus point this pobs won't be a bad approximation?
    float3 hit_point = prd->laser_hitpoint ; //
    float3 Lr = make_float3(0.0f);

    float3 wo = -ray_dir;

    float3 Ld = make_float3(0.f);

    if (params.num_lights > 0 && bsdfType != BSDFType::SPECULAR) {
        // Uniform sample light
        unsigned int sample_light_index = 0; // NLOS scenes assume there is only one light source in the scene
        Light l = params.lights[sample_light_index];
        LightSample ls;
        ls.hitpoint = hit_point;
        ls.wo = wo;
        ls.n = world_normal;

        // Compute direct lighting
        float3 dl = ComputeDirectLight(l, ls, mat, prd->depth);
        Ld = prd->contribution * dl;
      
   /*      if (fmaxf(Ld) < 0) {
            
            printf("Hit Point LS: (%f,%f,%f) | t: %f | contr: (%f,%f,%f) | dl: (%f,%f,%f), Ld: (%f,%f,%f)\n",  
                hit_point.x, hit_point.y, hit_point.z, optixGetRayTime(), 
                prd->contribution.x,prd->contribution.y,prd->contribution.z,
                dl.x,dl.y,dl.z,
                Ld.x,Ld.y,Ld.z);
         }*/
        prd->path_length += ls.dist;
        prd->Lr = Ld;
    }

}

static __device__ __inline__ void MITransientIntegrator(PerRayData_transientCamera* prd_camera,
                                                      const MaterialRecordParameters* mat_params,
                                                      unsigned int& material_id,
                                                      const unsigned int& num_blended_materials,
                                                      const float3& world_normal,
                                                      const float2& uv,
                                                      const float3& tangent,
                                                      const float& ray_dist,
                                                      const float3& ray_orig,
                                                      const float3& ray_dir) {
    // printf("TIMEGATED Integrator!\n");
    const MaterialParameters& mat = params.material_pool[material_id];
    BSDFType bsdfType = (BSDFType)mat.BSDFType;
    float3 hit_point = ray_orig + ray_dir * ray_dist;
   
    float3 L = make_float3(0.0f);

    float3 wo = -ray_dir;

    prd_camera->path_length += ray_dist;

    // Direct emission
    {

    }

    // Laser sampling
    float3 Lr = make_float3(0.f);
    if (params.nlos_laser_sampling) {
        /*printf("NLOS Laser Sampling: hp: (%f,%f,%f), lp: (%f,%f,%f)\n", 
            hit_point.x,hit_point.y,hit_point.z, 
            prd_camera->laser_focus_point.x,prd_camera->laser_focus_point.y,prd_camera->laser_focus_point.z);*/
        float laser_dist = Length(prd_camera->laser_focus_point - hit_point);
        float3 laser_dir = (prd_camera->laser_focus_point - hit_point) / laser_dist;
        // Shoot shadow ray to test visibility to laser target
        PerRayData_shadow prd_shadow = default_shadow_prd();
        prd_shadow.depth = prd_camera->depth + 1;
        prd_shadow.ramaining_dist = laser_dist;
        unsigned int opt1;
        unsigned int opt2;
        pointer_as_ints(&prd_shadow, opt1, opt2);
        unsigned int raytype = (unsigned int)SHADOW_RAY_TYPE;
        optixTrace(params.root, hit_point, laser_dir, params.scene_epsilon, laser_dist, optixGetRayTime(),
                   OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);

        float NdLaser = Dot(world_normal, -1*laser_dir);
      
        if (fmaxf(prd_shadow.attenuation) > 0 && laser_dist > 1e-8f) {  // 
           /* printf("NdLaser: %f, d: (%f,%f,%f), n: (%f,%f,%f)\n", NdLaser, laser_dir.x, laser_dir.y, laser_dir.z,
                   world_normal.x, world_normal.y, world_normal.z);*/
            BSDFSample laser_dir_bsdf;
            laser_dir_bsdf.wo = wo;
            laser_dir_bsdf.wi  = laser_dir;
            laser_dir_bsdf.n = world_normal;
            SampleBSDF(bsdfType,laser_dir_bsdf,mat,true);
            //laser_dir_bsdf.pdf = EvalBSDFPDF(bsdfType, wo, laser_dir, laser_dir_bsdf.n);
          
            if (NdLaser > 0) {
                float pdf_ls = (laser_dist * laser_dist) / NdLaser;
                laser_dir_bsdf.pdf /= pdf_ls;
            }
            // Shoot laser ray towards laser focus point
            PerRayData_laserSampleRay prd = default_laserSampleRay_prd();
            prd.sample_laser = true;
            prd.path_length = prd_camera->path_length + laser_dist;
            prd.bsdf_pdf = laser_dir_bsdf.pdf;
            prd.contribution = prd_camera->contrib_to_pixel * laser_dir_bsdf.pdf;
            prd.depth = prd_camera->depth + 1;
            prd.laser_hitpoint = prd_camera->laser_focus_point;
            unsigned int opt1;
            unsigned int opt2;
            pointer_as_ints(&prd, opt1, opt2);
            unsigned int raytype = (unsigned int)LASER_SAMPLE_RAY_TYPE;
            //printf("Hit Point: (%f,%f,%f)\n", hit_point.x, hit_point.y, hit_point.z);
        /*    printf("laser dist: %f t: %f | PL: %f |Target: (%f,%f,%f), o: (%f,%f,%f) d: (%f,%f,%f) | Proj HP: (%f,%f,%f)\n", 
                   laser_dist,
                   optixGetRayTime(),
                   prd.path_length,
                   prd.laser_hitpoint.x, prd.laser_hitpoint.y, prd.laser_hitpoint.z, 
                   hit_point.x, hit_point.y, hit_point.z, laser_dir.x,
                   laser_dir.y, laser_dir.z,
                   proj_hp.x, proj_hp.y, proj_hp.z);*/
              optixTrace(params.root, hit_point, laser_dir, params.scene_epsilon, laser_dist, laser_dist, OptixVisibilityMask(1),
                       OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT, 0, 1, 0, opt1, opt2, raytype);
         
            // Add transient sample
            if (fmaxf(prd.Lr) > 0) {
               /* printf("Laser Lr: (%f,%f,%f) | PL Before: %f, PL: %f\n", prd.Lr.x, prd.Lr.y, prd.Lr.z,
                         prd_camera->path_length + laser_dist, prd.path_length);*/
                TransientSample sample = {};
                sample.pathlength = prd.path_length;
                sample.color = prd.Lr;
                int idx = params.max_depth * prd_camera->current_pixel + (prd_camera->depth - 1);

                params.transient_buffer[idx] = sample;
                
            }
        }
    } 
    else { // Do standard NEE Direct lighting
        // Uniform sample light
        unsigned int sample_light_index =
            (unsigned int)(hiprand_uniform(&prd_camera->rng) *
                           params.num_lights);  // TODO: Won't work for whitted as no GI, have a global sampler instead?
        Light l = params.lights[sample_light_index];
        LightSample ls;
        ls.hitpoint = hit_point;
        ls.wo = wo;
        ls.n = world_normal;

        // Compute direct lighting
        Lr = prd_camera->contrib_to_pixel * ComputeDirectLight(l, ls, mat, prd_camera->depth);

        if (fmaxf(Lr) > 0) {
            TransientSample sample = {};
            sample.pathlength = prd_camera->path_length + ls.dist;
            sample.color = Lr;
            int idx = params.max_depth * prd_camera->current_pixel + (prd_camera->depth - 1);

            params.transient_buffer[idx] = sample;
        }
    }

    L += Lr;

    // Find next direction by either hidden geometry sampling or standard bsdf sampling
    {
        if (prd_camera->depth + 1 < params.max_depth) {
            BSDFSample sample;
            sample.wo = wo;
            sample.n = world_normal;
            float z1 = hiprand_uniform(&prd_camera->rng);
            float z2 = hiprand_uniform(&prd_camera->rng);
            SampleBSDF(bsdfType, sample, mat, false, z1, z2);

            if (luminance(sample.f) > params.importance_cutoff > 0 && sample.pdf > 0) {
                float NdL = Dot(sample.n, sample.wi);
                float3 next_contrib_to_pixel = prd_camera->contrib_to_pixel * sample.f * NdL / sample.pdf;
                if (fmaxf(next_contrib_to_pixel) < 0) {
                    prd_camera->depth_reached = prd_camera->depth;
                    return;
                }
                // Check possible rr termination
                float rr_thresh = .1f;
                if (fmaxf(next_contrib_to_pixel) < rr_thresh && prd_camera->depth > 3) {
                    float q = fmaxf((float).05, 1 - fmaxf(next_contrib_to_pixel));
                    float p = hiprand_uniform(&prd_camera->rng);
                    if (p < q) {
                        prd_camera->depth_reached = prd_camera->depth;
                        return;
                    }
                    next_contrib_to_pixel = next_contrib_to_pixel / (1 - q);
                }

                // Trace next ray
                PerRayData_transientCamera prd_reflection = default_transientCamera_prd(prd_camera->current_pixel);
                prd_reflection.integrator = prd_camera->integrator;
                prd_reflection.contrib_to_pixel = next_contrib_to_pixel;
                prd_reflection.rng = prd_camera->rng;
                prd_reflection.depth = prd_camera->depth + 1;
                prd_reflection.use_gi = prd_camera->use_gi;
                prd_reflection.path_length = prd_camera->path_length;
                prd_reflection.laser_focus_point = prd_camera->laser_focus_point;
                unsigned int opt1, opt2;
                pointer_as_ints(&prd_reflection, opt1, opt2);
                unsigned int raytype = (unsigned int)TRANSIENT_RAY_TYPE;
                optixTrace(params.root, hit_point, sample.wi, params.scene_epsilon, 1e16f, optixGetRayTime(),
                           OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);
                L += prd_reflection.color;
                prd_camera->depth_reached = prd_reflection.depth_reached;
            }
        } else {
            prd_camera->depth_reached = prd_camera->depth;
        }

        prd_camera->color += L;
        prd_camera->albedo = mat.Kd;  // Might change
        prd_camera->normal = world_normal;
    }
}

extern "C" __global__ void __closesthit__material_shader() {
    //printf("Material Shader!\n");
    // determine parameters that are shared across all ray types
    const MaterialRecordParameters* mat_params = (MaterialRecordParameters*)optixGetSbtDataPointer();

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = normalize(optixGetWorldRayDirection());  // this may be modified by the scaling transform
    const float ray_dist = optixGetRayTmax();

    float3 hit_point = ray_orig + ray_dir*ray_dist;

    //printf("NVDBVolShader: orig: (%f,%f,%f), dir:(%f,%f,%f)\n", ray_orig.x, ray_orig.y, ray_orig.z, ray_dir.x, ray_dir.y,ray_dir.z);
    float3 object_normal;
    float2 uv;
    float3 tangent;


    // check if we hit a triangle
    unsigned int material_id = mat_params->material_pool_id;
    const MaterialParameters& mat = params.material_pool[material_id];
    if (optixIsTriangleHit()) {
        GetTriangleData(object_normal, material_id, uv, tangent, mat_params->mesh_pool_id);
    } else {
        object_normal = make_float3(__int_as_float(optixGetAttribute_0()), __int_as_float(optixGetAttribute_1()),
                                    __int_as_float(optixGetAttribute_2()));
        uv = make_float2(__int_as_float(optixGetAttribute_3()), __int_as_float(optixGetAttribute_4()));
        tangent = make_float3(__int_as_float(optixGetAttribute_5()), __int_as_float(optixGetAttribute_6()),
                              __int_as_float(optixGetAttribute_7()));
    }

    

    if (mat.kn_tex) {
        float3 bitangent = normalize(Cross(object_normal, tangent));
        const float4 tex = tex2D<float4>(mat.kn_tex, uv.x*mat.tex_scale.x, uv.y*mat.tex_scale.y);
        float3 normal_delta = make_float3(tex.x, tex.y, tex.z) * 2.f - make_float3(1.f);
        object_normal =normalize(normal_delta.x * tangent + normal_delta.y * bitangent + normal_delta.z * object_normal);
    }

    float3 world_normal = normalize(optixTransformNormalFromObjectToWorldSpace(object_normal));

    // from here on out, things are specific to the ray type
    RayType raytype = (RayType)optixGetPayload_2();


    switch (raytype) {
        case CAMERA_RAY_TYPE:
            PerRayData_camera* prd_cam = getCameraPRD();
            switch (prd_cam->integrator)
            {
                case Integrator::PATH:
                    CameraPathIntegrator(prd_cam, mat_params, material_id, mat_params->num_blended_materials,
                                 world_normal, uv, tangent, ray_dist, ray_orig, ray_dir);
                    break;
                case Integrator::VOLUMETRIC:
                    break; 
                case Integrator::LEGACY:
                    //printf("LEGACY SHADER!\n");
                    CameraShader(prd_cam, mat_params, material_id, mat_params->num_blended_materials,world_normal, uv,tangent, ray_dist, ray_orig, ray_dir);
                    break;
                default:
                    break;
            }
            // switch(mat.BSDFType) {
            //     case 0:
            //         CameraShader(getCameraPRD(), mat_params, material_id, mat_params->num_blended_materials, world_normal, uv,
            //                 tangent, ray_dist, ray_orig, ray_dir);
            //         break;
            //     case 1:
            //         CameraHapkeShader(getCameraPRD(), mat_params, material_id, mat_params->num_blended_materials, world_normal, uv,
            //                 tangent, ray_dist, ray_orig, ray_dir);
            //         break;
            //     case 2:
            //         CameraDiffuseShader(getCameraPRD(), mat_params, material_id, mat_params->num_blended_materials, world_normal, uv,
            //                 tangent, ray_dist, ray_orig, ray_dir);
            //         break;
            //     case 3:
            //         CameraVolumetricShader(getCameraPRD(), mat_params, material_id, mat_params->num_blended_materials,
            //                             world_normal, uv, tangent, ray_dist, ray_orig, ray_dir);
            //         break;
            //     default:
            //         break;
            // }
                
            break;
        case TRANSIENT_RAY_TYPE:
            PerRayData_transientCamera* transCam_prd = getTransientCameraPRD();
            switch (transCam_prd->integrator)
            {
                case Integrator::TRANSIENT:
                    TransientPathIntegrator(transCam_prd, mat_params, material_id, mat_params->num_blended_materials, world_normal, uv,
                        tangent, ray_dist, ray_orig, ray_dir);
                    break;
                case Integrator::TIMEGATED:
                   TimeGatedIntegrator(transCam_prd, mat_params, material_id, mat_params->num_blended_materials,
                                            world_normal, uv, tangent, ray_dist, ray_orig, ray_dir);
                   break;
                case Integrator::MITRANSIENT:
                   MITransientIntegrator(transCam_prd, mat_params, material_id, mat_params->num_blended_materials,
                                        world_normal, uv, tangent, ray_dist, ray_orig, ray_dir);
                    break;
                default:
                    break;
            }
            break;
        case LIDAR_RAY_TYPE:
            LidarShader(getLidarPRD(), mat, world_normal, uv, tangent, ray_dist, ray_orig, ray_dir);
            break;
        case RADAR_RAY_TYPE:
            RadarShader(getRadarPRD(), mat, world_normal, uv, tangent, ray_dist, ray_orig, ray_dir,
                        mat_params->translational_velocity, mat_params->angular_velocity, mat_params->objectId);
            break;
        case SHADOW_RAY_TYPE:
            ShadowShader(getShadowPRD(), mat, world_normal, uv, tangent, ray_dist, ray_orig, ray_dir);
            break;
        case SEGMENTATION_RAY_TYPE:
            SemanticShader(getSemanticPRD(), mat, world_normal, uv, tangent, ray_dist, ray_orig, ray_dir);
            break;
        case DEPTH_RAY_TYPE:
            DepthShader(getDepthCameraPRD(), mat, world_normal, uv, tangent, ray_dist, ray_orig, ray_dir);
            break;
        case LASER_SAMPLE_RAY_TYPE:
            PerRayData_laserSampleRay* prd = getLaserPRD();
            if (prd->sample_laser) {
                LaserNEEE(prd, mat_params, material_id, mat_params->num_blended_materials,
                                      world_normal, uv, tangent, ray_dist, ray_orig, ray_dir);
            } else {
                prd->laser_hitpoint = hit_point;
            }
            break;
    }
}
